#include "hip/hip_runtime.h"
// Copyright (c) 2022, Raymond Leung
// All rights reserved.
//
// This source code is licensed under the BSD-3-clause license found
// in the LICENSE.md file in the root directory of this source tree.
//
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <iterator>
#include <typeinfo>
#include <vector>

#include <stdint.h>
#include "bvh_structure.h"
#include "rsi_geometry.h"

using namespace std;
using namespace lib_bvh;
using namespace lib_rsi;

//-------------------------------------------------
// This implementation corresponds to version v3
// with support for barycentric mode and the
// intercept_count experimental feature
//-------------------------------------------------

static void HandleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

static void CheckSyncAsyncErrors(const char* file, int line) {
    // Inspired from https://developer.nvidia.com/blog/how-query-device-properties-and-handle-errors-cuda-cc/
    hipError_t errSync =
        hipGetLastError();  // returns the value of the latest asynchronous error and also resets it to hipSuccess.
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess) {
        printf("Sync kernel error\n");
        HandleError(errSync, file, line);
    }
    if (errAsync != hipSuccess) {
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
        HandleError(errAsync, file, line);
    }
}

#define CUDA_SYNCHRO_CHECK() (CheckSyncAsyncErrors(__FILE__, __LINE__))

template <class T>
int readData(string fname, vector<T>& v, int dim = 1, bool silent = false) {
    ifstream infile(fname.c_str(), ios::binary | ios::ate);
    if (!infile) {
        cerr << "File " << fname << " not found" << endl;
        exit(1);
    }
    ifstream::pos_type nbytes = infile.tellg();
    infile.seekg(0, infile.beg);
    const int elements = nbytes / sizeof(T);
    v.resize(elements);
    infile.read(reinterpret_cast<char*>(v.data()), nbytes);
    if (!silent) {
        cout << fname << " contains " << nbytes << " bytes, " << v.size() << " <" << typeid(v.front()).name() << ">, "
             << v.size() / dim << " elements" << endl;
    }
    return elements / dim;
}

template <class T>
void writeData(string fname, vector<T>& v) {
    ofstream outfile(fname.c_str(), ios::out | ios::binary);
    if (!outfile) {
        cerr << "Cannot create " << fname << " for writing" << endl;
        exit(1);
    }
    outfile.write(reinterpret_cast<char*>(v.data()), v.size() * sizeof(T));
    outfile.close();
}

// =================== JSON added helper function =======================
void readMeshData(string fname, vector<float>& v, vector<int>& surf) {
    std::ifstream file(fname);
    std::string line;

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::string prefix;
        iss >> prefix;

        if (prefix == "v") {
            float x,y,z;
            iss >> x >> y >> z;
            v.push_back(x);
            v.push_back(y);
            v.push_back(z);
        } else if (prefix == "f") {
            std::string vertex;
            for (int i = 0; i < 3; ++i) {
                iss >> vertex;
                size_t slash_pos = vertex.find('/');
                if (slash_pos != std::string::npos) {
                    vertex = vertex.substr(0, slash_pos);
                }
                int idx = std::stoi(vertex) - 1;
                surf.push_back(idx);
            }
        }
    }

}

// ======================================================================
// this helper converts intersection points from Baycentric coordinates to Cartesian coordinates
void intersect_helper(vector<int>& h_intersectTriangle,
    vector<float>& h_baryT, vector<float>& h_baryU, vector<float>& h_baryV, const int& nVertices, const int& nTriangles, const int& nRays, vector<float>& h_vertices, vector<int>& h_triangles, vector<float>& p_intersect){
        for(int i = 0; i < nRays; i++){
            if(h_intersectTriangle[i] != -1){
                int f = h_intersectTriangle[i];
                int i0 = h_triangles[3*f];
                int i1 = h_triangles[3*f+1];
                int i2 = h_triangles[3*f+2];
                float t = h_baryT[i];
                float u = h_baryU[i];
                float v = h_baryV[i];
                float x = (1-u-v)*h_vertices[3*i0] + u*h_vertices[3*i1] + v*h_vertices[3*i2];
                float y = (1-u-v)*h_vertices[3*i0+1] + u*h_vertices[3*i1+1] + v*h_vertices[3*i2+1];
                float z = (1-u-v)*h_vertices[3*i0+2] + u*h_vertices[3*i1+2] + v*h_vertices[3*i2+2];
                p_intersect[3*i] = x;
                p_intersect[3*i+1] = y;
                p_intersect[3*i+2] = z;
            }
        }
    }  

void ray_generation_helper(vector<float>& h_rayFrom, vector<float>& h_rayTo, int& nRays){
    // h_rayFrom = {0.0, 0.0, -0.4, 10.0, 0.0, 0.0};
    // h_rayTo = {0.0, 0.0, 5.0, 10.0, 0.0, 10.0};
    // nRays = 2;

    //a dummy ray generation of SCM grid 1m x 0.5 m, centered at 0, with mesh resolution 0.01.
    float x_dim = 2.0; float y_dim = 1.0;
    float x_c = 0.0; float y_c = 0.0;
    float resolution = 0.01;
    int n_x = int(x_dim / resolution)+1;
    int n_y = int(y_dim / resolution)+1;
    nRays = n_x * n_y;

    for(int i = 0; i < n_x; i++){
        for(int j = 0; j < n_y; j++){
            h_rayFrom.push_back(x_c - x_dim/2 + i*resolution);
            h_rayFrom.push_back(y_c - y_dim/2 + j*resolution);
            h_rayFrom.push_back(-0.4);
            h_rayTo.push_back(x_c - x_dim/2 + i*resolution);
            h_rayTo.push_back(y_c - y_dim/2 + j*resolution);
            h_rayTo.push_back(5.0);
        }
    }
}

// ======================================================================

int main(int argc, char* argv[]) {
    const bool checkEnabled(true);
    const float largePosVal(2.5e+8);
    vector<float> h_vertices;
    vector<int> h_triangles;
    vector<float> h_rayFrom;
    vector<float> h_rayTo;
    vector<int> h_crossingDetected;
    vector<int> h_intersectTriangle;
    vector<float> h_baryT, h_baryU, h_baryV;
    int nVertices, nTriangles, nRays;

    bool quietMode=false;
    /*
    Ray-surface intersection results are reported as follows:
      barycentric = false
      |  if interceptsCount is false (by default)
      |     return boolean array, h_crossingDetected[r] is set to 0 or 1
      |  else report the number of surface intersections for each ray
      |     return integer array, h_crossingDetected[r] >= 0
      barycentric = true
      |  return index of intersecting triangle (f) via h_intersectTriangle[r]
      |  (-1 if none) and the intersecting point P via barycentric coordinates
      |  (t[r], u[r], v[r]) where t = distance(rayFrom, surface), P =
      |  (1-u-v)*V[0] + u*V[1] + v*V[2], V[i] = vertices[triangles[f][i]].
    */
    bool barycentric = true;
    bool interceptsCount(argc > 6 ? strcmp(argv[6], "intercept_count") == 0 : false);

    // read input data into host memory
    readMeshData("cobra_wheel.obj", h_vertices, h_triangles);
    nVertices = h_vertices.size() / 3;
    nTriangles = h_triangles.size() / 3;
    cout << "v size:"<< nVertices << " t size:" << nTriangles << endl;
    //nVertices = readData(fileVertices, h_vertices, 3, quietMode);
    //nTriangles = readData(fileTriangles, h_triangles, 3, quietMode);

    if (h_triangles.size() == 3) {
        // Add an extra triangle so that BVH traversal works in an
        // uncomplicated way without throwing an exception. It
        // expects at least one split node at the top of the binary
        // radix tree where the left and right child nodes are defined.
        for (int i = 0; i < 3; i++)
            h_triangles.push_back(0);
        nTriangles += 1;
    }

    //nRays = readData(fileFrom, h_rayFrom, 3, quietMode);
    //int nRaysTo = readData(fileTo, h_rayTo, 3, quietMode);
    //assert(nRaysTo == nRays);

    ray_generation_helper(h_rayFrom, h_rayTo, nRays);
    std::cout << "nRays: " << nRays << std::endl;

    h_crossingDetected.resize(nRays);

    hipEvent_t start, end;
    float time = 0;
    float *d_vertices, *d_rayFrom, *d_rayTo;
    int *d_triangles, *d_crossingDetected, *d_intersectTriangle;
    float *d_baryT, *d_baryU, *d_baryV;
    AABB* d_rayBox;
    int sz_vertices(3 * nVertices * sizeof(float)), sz_triangles(3 * nTriangles * sizeof(int)),
        sz_rays(3 * nRays * sizeof(float)), sz_rbox(nRays * sizeof(AABB)), sz_id(nRays * sizeof(int)),
        sz_bary(nRays * sizeof(float));
    HANDLE_ERROR(hipMalloc(&d_vertices, sz_vertices));
    HANDLE_ERROR(hipMalloc(&d_triangles, sz_triangles));
    HANDLE_ERROR(hipMalloc(&d_rayFrom, sz_rays));
    HANDLE_ERROR(hipMalloc(&d_rayTo, sz_rays));
    HANDLE_ERROR(hipMalloc(&d_rayBox, sz_rbox));

    if (!barycentric) {
        HANDLE_ERROR(hipMalloc(&d_crossingDetected, sz_id));
        HANDLE_ERROR(hipMemset(d_crossingDetected, 0, sz_id));
    } else {
        h_intersectTriangle.resize(nRays);
        h_baryT.resize(nRays);
        h_baryU.resize(nRays);
        h_baryV.resize(nRays);
        HANDLE_ERROR(hipMalloc(&d_intersectTriangle, sz_id));
        HANDLE_ERROR(hipMalloc(&d_baryT, sz_bary));
        HANDLE_ERROR(hipMalloc(&d_baryU, sz_bary));
        HANDLE_ERROR(hipMalloc(&d_baryV, sz_bary));
    }
    HANDLE_ERROR(hipMemcpy(d_vertices, h_vertices.data(), sz_vertices, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_triangles, h_triangles.data(), sz_triangles, hipMemcpyHostToDevice));

    // grid partitions
    int blockX = 1024, gridXr = (int)ceil((float)nRays / blockX), gridXt = (int)ceil((float)nTriangles / blockX),
        gridXLambda = 16;  // N_{grids}
    if (!quietMode) {
        cout << blockX << " threads/block, grids: {triangles: " << gridXt << ", rays: " << gridXLambda << "}" << endl;
    }
    float minval[3], maxval[3], half_delta[3], inv_delta[3];
    vector<uint64_t> h_morton;
    vector<int> h_sortedTriangleIDs;

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&end));
    HANDLE_ERROR(hipEventRecord(start));
    HANDLE_ERROR(hipMemcpy(d_rayFrom, h_rayFrom.data(), sz_rays, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_rayTo, h_rayTo.data(), sz_rays, hipMemcpyHostToDevice));

    // initialise arrays
    if (barycentric) {
        initArrayKernel<<<gridXr, blockX>>>(d_intersectTriangle, -1, nRays);
        initArrayKernel<<<gridXr, blockX>>>(d_baryT, largePosVal, nRays);
    }
    HANDLE_ERROR(hipDeviceSynchronize());

    // compute ray-segment bounding boxes
    rbxKernel<<<gridXr, blockX>>>(d_rayFrom, d_rayTo, d_rayBox, nRays);
    HANDLE_ERROR(hipDeviceSynchronize());

    // order triangles using Morton code
    //- normalise surface vertices to canvas coords
    getMinMaxExtentOfSurface<float>(h_vertices, minval, maxval, half_delta, inv_delta, nVertices, quietMode);
    //- convert centroid of triangles to morton code
    createMortonCode<float, uint64_t>(h_vertices, h_triangles, minval, half_delta, inv_delta, h_morton, nTriangles);
    //- sort before constructing binary radix tree
    sortMortonCode<uint64_t>(h_morton, h_sortedTriangleIDs);
    if (!quietMode && checkEnabled) {
        cout << "checking sortMortonCode" << endl;
        for (int j = 0; j < min(12, nTriangles); j++) {
            cout << j << ": (" << h_sortedTriangleIDs[j] << ") " << h_morton[j] << endl;
        }
    }
    // data structures used in agglomerative LBVH construction
    BVHNode *d_leafNodes, *d_internalNodes;
    uint64_t* d_morton;
    int* d_sortedTriangleIDs;
    CollisionList* d_hitIDs;
    int sz_morton(nTriangles * sizeof(uint64_t)), sz_sortedIDs(nTriangles * sizeof(int)),
        sz_hitIDs(gridXLambda * blockX * sizeof(CollisionList));
    InterceptDistances* d_interceptDists;
    int sz_interceptDists(gridXLambda * blockX * sizeof(InterceptDistances));
    HANDLE_ERROR(hipMalloc(&d_leafNodes, nTriangles * sizeof(BVHNode)));
    HANDLE_ERROR(hipMalloc(&d_internalNodes, nTriangles * sizeof(BVHNode)));
    HANDLE_ERROR(hipMalloc(&d_morton, sz_morton));
    HANDLE_ERROR(hipMalloc(&d_sortedTriangleIDs, sz_sortedIDs));
    HANDLE_ERROR(hipMalloc(&d_hitIDs, sz_hitIDs));
    if (interceptsCount) {
        HANDLE_ERROR(hipMalloc(&d_interceptDists, sz_interceptDists));
    }
    HANDLE_ERROR(hipMemcpy(d_morton, h_morton.data(), sz_morton, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d_sortedTriangleIDs, h_sortedTriangleIDs.data(), sz_sortedIDs, hipMemcpyHostToDevice));
    std::vector<uint64_t>().swap(h_morton);
    std::vector<int>().swap(h_sortedTriangleIDs);

    bvhResetKernel<<<gridXt, blockX>>>(d_vertices, d_triangles, d_internalNodes, d_leafNodes, d_sortedTriangleIDs,
                                       nTriangles);
    HANDLE_ERROR(hipDeviceSynchronize());

    bvhConstruct<uint64_t><<<gridXt, blockX>>>(d_internalNodes, d_leafNodes, d_morton, nTriangles);
    // HANDLE_ERROR(hipDeviceSynchronize());
    CUDA_SYNCHRO_CHECK();

    if (barycentric) {
        bvhIntersectionKernel<<<gridXLambda, blockX>>>(d_vertices, d_triangles, d_rayFrom, d_rayTo, d_internalNodes,
                                                       d_rayBox, d_hitIDs, d_intersectTriangle, d_baryT, d_baryU,
                                                       d_baryV, nTriangles, nRays);
    } else if (interceptsCount) {
        bvhIntersectionKernel<<<gridXLambda, blockX>>>(d_vertices, d_triangles, d_rayFrom, d_rayTo, d_internalNodes,
                                                       d_rayBox, d_hitIDs, d_interceptDists, d_crossingDetected,
                                                       nTriangles, nRays);
    } else {
        bvhIntersectionKernel<<<gridXLambda, blockX>>>(d_vertices, d_triangles, d_rayFrom, d_rayTo, d_internalNodes,
                                                       d_rayBox, d_hitIDs, d_crossingDetected, nTriangles, nRays);
    }
    HANDLE_ERROR(hipDeviceSynchronize());

    HANDLE_ERROR(hipEventRecord(end));
    HANDLE_ERROR(hipEventSynchronize(end));
    HANDLE_ERROR(hipEventElapsedTime(&time, start, end));

    if (!barycentric) {
        HANDLE_ERROR(hipMemcpy(h_crossingDetected.data(), d_crossingDetected, sz_id, hipMemcpyDeviceToHost));
        writeData("results_i32", h_crossingDetected);
    } else {
        HANDLE_ERROR(hipMemcpy(h_intersectTriangle.data(), d_intersectTriangle, sz_id, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(h_baryT.data(), d_baryT, sz_bary, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(h_baryU.data(), d_baryU, sz_bary, hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(h_baryV.data(), d_baryV, sz_bary, hipMemcpyDeviceToHost));
        writeData("intersectTriangle_i32", h_intersectTriangle);
        writeData("barycentricT_f32", h_baryT);
        writeData("barycentricU_f32", h_baryU);
        writeData("barycentricV_f32", h_baryV);
    }

    vector<float> p_intersect(nRays * 3, -1.0);
    // calculate the intersection point based on the barycentric coordinates
    intersect_helper(h_intersectTriangle,h_baryT, h_baryU, h_baryV, nVertices, nTriangles, nRays, h_vertices, h_triangles, p_intersect);

    // sanity check
    vector<int>& outcome = !barycentric ? h_crossingDetected : h_intersectTriangle;
    if (!quietMode) {
        cout << "Results for all intersection elements:" << endl;
        for (int i = 0; i < nRays; i++) {
            if(outcome[i] != -1){
                cout << i << ": " << outcome[i] << "," << p_intersect[3*i] << "," << p_intersect[3*i+1] << "," << p_intersect[3*i+2] << endl;
            }

        }
        cout << "Processing time: ";
        cout << time << " ms" << endl;
    }
}
